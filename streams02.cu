
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void omg_gpu(float* array) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < 25; ++i)
        array[index] = 1.5 * array[index] * (1- array[index]);
}

int main() {
    int elements = 1 << 28;

    float* host;
    hipHostMalloc(&host, sizeof(float) * elements, hipHostMallocDefault);
    for(int i = 0; i < elements; ++i) {
        host[i] = (1.f * i) / elements;
    }

    float* device;
    hipMalloc(&device, sizeof(float) * elements);

    dim3 block(128);
    dim3 grid(elements / block.x);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(device, host, sizeof(float) * elements, hipMemcpyDefault, stream);

    omg_gpu<<<grid, block, 0, stream>>>(device);

    hipMemcpyAsync(host, device, sizeof(float) * elements, hipMemcpyDefault, stream);

    hipStreamSynchronize(stream);

    hipStreamDestroy(stream);
    hipFree(device);
    hipHostFree(host);

    return 0;
}