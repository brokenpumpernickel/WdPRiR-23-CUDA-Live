
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void omg_gpu(float* array) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < 25; ++i)
        array[index] = 1.5 * array[index] * (1- array[index]);
}

int main() {
    int elements = 1 << 25;

    float* host;
    hipHostMalloc(&host, sizeof(float) * elements, hipHostMallocDefault);
    for(int i = 0; i < elements; ++i) {
        host[i] = (1.f * i) / elements;
    }

    float* device;
    hipMalloc(&device, sizeof(float) * elements);

    int nstreams = 4;
    hipStream_t* streams = (hipStream_t*) malloc(sizeof(hipStream_t) * nstreams);
    for(int i = 0; i < nstreams; ++i)
        hipStreamCreate(&streams[i]);

    int stream_elements = elements / nstreams;
    dim3 block(128);
    dim3 grid(stream_elements / block.x);

    for(int i = 0; i < nstreams; ++i) {
        hipMemcpyAsync(&device[i * stream_elements], &host[i * stream_elements], sizeof(float) * stream_elements, hipMemcpyDefault, streams[i]);
    }

    for(int i = 0; i < nstreams; ++i) {
        omg_gpu<<<grid, block, 0, streams[i]>>>(&device[i * stream_elements]);
    }

    for(int i = 0; i < nstreams; ++i) {
        hipMemcpyAsync(&host[i * stream_elements], &device[i * stream_elements], sizeof(float) * stream_elements, hipMemcpyDefault, streams[i]);
    }

    for(int i = 0; i < nstreams; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipFree(device);
    hipHostFree(host);

    return 0;
}