
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void omg_gpu(float* array) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < 25; ++i)
        array[index] = 1.5 * array[index] * (1 - array[index]);
}

int main() {
    int elements = 1 << 28;

    float* host;
    hipHostMalloc(&host, sizeof(float) * elements, hipHostMallocDefault);
    for(int i = 0; i < elements; ++i) {
        host[i] = (1.f * i) / elements;
    }

    float* device;
    hipMalloc(&device, sizeof(float) * elements);

    dim3 block(128);
    dim3 grid(elements / block.x);

    hipMemcpy(device, host, sizeof(float) * elements, hipMemcpyDefault);

    omg_gpu<<<grid, block>>>(device);

    hipMemcpy(host, device, sizeof(float) * elements, hipMemcpyDefault);

    hipFree(device);
    hipHostFree(host);

    return 0;
}